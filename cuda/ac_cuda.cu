
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <stdint.h>

//#include "../backend.h"

/* Device */
__inline__ __device__ int get_index(){
    return blockIdx.x * blockDim.x + threadIdx.x;
}

__inline__ __device__ int4 get_neighborhood(int index, int width, int length){
	int4 neighbors;

	int col = index % width;
	
	//left
	neighbors.x = col == 0 ? index + (width - 1) : index - 1;
	//up
	neighbors.y = index < width ? length - (width - index) : index - width; 
	//right
	neighbors.z = col == (width - 1) ? index - (width - 1) : index + 1;
	//down
	neighbors.w = (index + width) >= length ? col /* idx mod width */ : index + width;

	return neighbors;
}

__global__ void parity_automaton(uint* src, uint*dst, int width, int length){
	// Recover the index
	int index = get_index();

	// get direct neighbors of the pixel
	int4 neighbors = get_neighborhood(index, width, length);
	
	dst[index] = src[neighbors.x] ^ src[neighbors.y] ^ src[neighbors.z] ^ src[neighbors.w];
}

// cyclic next state function
__inline__ __device__ uint cyclic(uint center, uint left, uint up, uint right, uint down, uint max){
	uint k1 = (center + 1) % (max + 1);
	return (left == k1) || (up == k1) || (right == k1) || (down == k1) ? k1 : center;
}

__global__ void cyclic_automaton(uint *src, uint *dst, int width, int length, uint max_val) {
	// Recover the index
	int index = get_index();

	// get direct neighbors of the pixel
	int4 neighbors = get_neighborhood(index, width, length);

	// set the pixel value in destination matrix
	dst[index] = cyclic(src[index], src[neighbors.x], src[neighbors.y], src[neighbors.z], src[neighbors.w], max_val);
}

// Host
#define THREADS_PER_BLOCK 64

static uint *d_src = NULL;
static uint *d_dst = NULL;
static int32_t d_width = 0;
static int32_t d_length = 0;
static bool d_parity = true;
static uint32_t d_max_value = 0;
static int32_t size = 0;

extern "C" {
	void init(){}
	void set_args(bool parity, uint32_t *matrix, int32_t width, int32_t length, uint32_t max_value){
		d_parity = parity;
		d_width = width;
		d_length = length;
		d_max_value = max_value;
		size = length * sizeof(*matrix);
		
		if(d_src){
			hipFree(d_src);
		}
		if(d_dst){
			hipFree(d_dst);
		}
		/* allocate space for device copies src and dst */
		hipMalloc((void **) &d_src, size);
		hipMalloc((void **) &d_dst, size);
		/* copy src to device */
		hipMemcpy(d_src, matrix, size, hipMemcpyHostToDevice);
	}
	void iterate(uint32_t iteration){
		/* launch the kernel on the GPU */
		/* insert the launch parameters to launch the kernel properly using blocks and threads */ 
		uint* tmp;
		#pragma unroll 4
		for(int i = 0; i < iteration; i++){
			if(d_parity){
				parity_automaton<<<(d_length + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_src, d_dst, d_width, d_length);
			}else{
				cyclic_automaton<<<(d_length + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_src, d_dst, d_width, d_length, d_max_value);
			}
			// Swap dst and src
			tmp = d_src;
			d_src = d_dst;
			d_dst = tmp; 
		}
		// At the end the final result is in d_src
	}
	void get_result(uint32_t *matrix){
		/* copy src back to host */
		hipMemcpy(matrix, d_src, size, hipMemcpyDeviceToHost);
	}
	void destroy(){
		hipFree(d_src);
		hipFree(d_dst);
		d_src = d_dst = NULL;
	}
}