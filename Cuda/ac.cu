
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <string.h>

/* Device */
__inline__ __device__ int get_index(){
    return blockIdx.x * blockDim.x + threadIdx.x;
}

__inline__ __device__ int4 get_neighborhood(int index, int width, int length){
	int4 neighbors;

	int col = index % width;
	
	//left
	neighbors.x = col == 0 ? index + (width - 1) : index - 1;
	//up
	neighbors.y = index < width ? length - (width - index) : index - width; 
	//right
	neighbors.z = col == (width - 1) ? index - (width - 1) : index + 1;
	//down
	neighbors.w = (index + width) >= length ? col /* idx mod width */ : index + width;

	return neighbors;
}

__global__ void parity_automaton(uint* src, uint*dst, int width, int length){
	int index = get_index();
	int4 neighbors = get_neighborhood(index, width, length);
	
	dst[index] = src[neighbors.x] ^ src[neighbors.y] ^ src[neighbors.z] ^ src[neighbors.w];
}

// Host
#define THREADS_PER_BLOCK 64

int main(int argc, const char* argv[])
{
	int iteration = 1000, width = 1000, height = 1000;
    if (argc > 1)
    {
        iteration = atoi(argv[1]);
    }else{
        printf("Usage:\n    %s iteration [width] [height]\n", argv[0]);
        exit(1);
    }
    
    if(argc > 2){
        width = atoi(argv[2]);
        height = width;
    }
    else if (argc > 3)
    {
        width = atoi(argv[2]);
        height = atoi(argv[3]);
	}
	
	int length = width * height;

	uint *src, *dst;
	uint *d_src, *d_dst;
	int size = length * sizeof(uint);

	/* allocate space for device copies src and dst */
	hipMalloc((void **) &d_src, size);
	hipMalloc((void **) &d_dst, size);

	/* allocate space for host copies of src and dst and setup input values */
	src = (uint *)malloc(size);
	dst = (uint *)malloc(size);

	for(int i = 0; i < length; i++)
	{
		src[i] = i % 2;
	}

	/* copy src to device */
	hipMemcpy(d_src, src, size, hipMemcpyHostToDevice);

	struct timespec start, finish;
    double seconds_elapsed = 0.0;

    clock_gettime(CLOCK_MONOTONIC, &start);
	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */ 
	uint* tmp;
	for(int i = 0; i < iteration; i++){
		parity_automaton<<<(length + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_src, d_dst, width, length);
		// Swap dst and src
		tmp = d_src;
		d_src = d_dst;
		d_dst = tmp; 
	}
	// At the end the final result is in d_src

	/* copy src back to host */
	hipMemcpy(src, d_src, size, hipMemcpyDeviceToHost);
	clock_gettime(CLOCK_MONOTONIC, &finish);
    seconds_elapsed += (double)(finish.tv_sec - start.tv_sec) + (finish.tv_nsec - start.tv_nsec) / 1.0e9;
    printf("Result: %lf\n", seconds_elapsed);

	/* clean up */
	free(src);
	free(dst);
	hipFree(d_src);
	hipFree(d_dst);
	
	return EXIT_SUCCESS;
}